#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"  
#include ""  

__global__ void Parallel(const int* a, const int* b, int* c)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = b[i] + a[i];
}

extern "C" void addKernel(const int* h_a, const int* h_b, int* h_c, int size)
{
	int* d_A, * d_B, * d_C;
	hipSetDevice(0);
	hipMalloc((void**)&d_A,  size);
	hipMalloc((void**)&d_B,  size);
	hipMalloc((void**)&d_C,  size);
	hipMemcpy(d_A, h_a,  size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_b,  size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_c,  size, hipMemcpyHostToDevice);

	Parallel << <2, 128 >> > (d_A, d_B, d_C);

	hipMemcpy(h_c, d_C, size , hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}